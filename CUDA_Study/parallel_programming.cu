#include "hip/hip_runtime.h"
#include "common/book.h"
#define N 10

__global__ void add(int *a, int *b, int *c){
  int tid = blockIdx.x;
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}

int main()
{ 
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  // GPU 分配内存
  HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

  // CPU上操作数据赋值工作
  for(int i = 0; i < N; ++i){
    a[i] = -i;
    b[i] = i * i;
  }

  // 将数组a和b赋值到GPU中
  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  add<<<N, 1>>>(dev_a, dev_b, dev_c);

  HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
  
  for(int i = 0; i < N; ++i)
    printf("%d + %d = %d\n", a[i], b[i], c[i]);

  hipFree(dev_a);hipFree(dev_b);hipFree(dev_c);
  return 0;
}
